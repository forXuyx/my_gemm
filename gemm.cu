#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "common.h"


// gpu版本的矩阵乘法（naive版）
__global__ void naiveGemm (float *a, float *b, float *c, const int M, const int N, const int K) {

    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    if (m < M && n < N) {
        float sum = 0;
        #pragma unroll
        for (int k = 0; k < K; k ++ ) {
            sum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
        }
        c[OFFSET(m, n, N)] = sum;
    }
}

int main() {

    // 矩阵大小
    const int M_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int N_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int K_list[15] = {1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024};
    const int outer_repeat = 10, inner_repeat = 1;
    const int M = 512, N = 512, K = 512; // 矩阵大小
    const int BM = 32, BN = 32; // 块大小
    const int TESTNUM = 15; // 测试次数

    // --------------------------------------------
    // cublas版本的矩阵乘法

    printf("\nKernel = cublasGemm\n");

    // 计算误差
    float max_error = testCublasMaxError(M, N, K);
    printf("Max error: %f\n", max_error);

    // 测试性能
    for (int i = 0; i < TESTNUM; i ++ ) {
        const int M = M_list[i], N = N_list[i], K = K_list[i];

        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int j = 0; j < outer_repeat; j ++ ) {
            double this_sec = testCublasPerformance(M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        double avg_Gflops = 2.0 * M * N * K / avg_sec / 1024 / 1024 / 1024;

        printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
    }

    // --------------------------------------------
    // naive版本的矩阵乘法

    printf("\nKernel = naiveGemm\n");
    void (*gpuGemm) (float *, float *, float *, const int, const int, const int) = naiveGemm;

    // 计算误差
    dim3 blk(BM, BN);
    dim3 grid(ceil_div(M, BM), ceil_div(N, BN));
    max_error = testMaxError(gpuGemm, grid, blk, M, N, K);
    printf("Max error: %f\n", max_error);

    // 测试性能
    for (int i = 0; i < TESTNUM; i ++ ) {
        const int M = M_list[i], N = N_list[i], K = K_list[i];

        dim3 blk(BM, BN);
        dim3 grid(ceil_div(M, BM), ceil_div(N, BN));

        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int j = 0; j < outer_repeat; j ++ ) {
            double this_sec = testPerformance(gpuGemm, grid, blk, M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        double avg_Gflops = 2.0 * M * N * K / avg_sec / 1024 / 1024 / 1024;

        printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
    }

    // --------------------------------------------


    return 0;
}